#include "hip/hip_runtime.h"
#include "parser.h"

__constant__ int initialZigzag[64]; 
__device__ int global_sync_flag;
__device__ int counter = 0;


__device__ int clip(int value) {
    if (value < -256) return -256;
    if (value > 255) return 255;
    return value;
}

__device__ void idctRow(int* block) {
    int x0, x1, x2, x3, x4, x5, x6, x7;

    // Shortcut: if all AC terms are zero, directly scale the DC term
    if (!((x1 = block[4]<<11) | (x2 = block[6]) | (x3 = block[2]) | (x4 = block[1]) | (x5 = block[7]) | (x6 = block[5]) | (x7 = block[3]))) {
        block[0] = block[1] = block[2] = block[3] = block[4] = block[5] = block[6] = block[7] = block[0]<<3;
        return;
    }
    // Scale the DC coefficient
    x0 = (block[0]<<11) + 128;

    int x8 = C7 * (x4 + x5);
    x4 = x8 + (C1 - C7) * x4;
    x5 = x8 - (C1 + C7) * x5;
    x8 = C3 * (x6 + x7);
    x6 = x8 - (C3 - C5) * x6;
    x7 = x8 - (C3 + C5) * x7;

    x8 = x0 + x1;
    x0 -= x1;
    x1 = C6 * (x3 + x2);
    x2 = x1 - (C2 + C6) * x2;
    x3 = x1 + (C2 - C6) * x3;
    x1 = x4 + x6;
    x4 -= x6;
    x6 = x5 + x7;
    x5 -= x7;

    x7 = x8 + x3;
    x8 -= x3;
    x3 = x0 + x2;
    x0 -= x2;
    x2 = (181 * (x4 + x5) + 128) >> 8;
    x4 = (181 * (x4 - x5) + 128) >> 8;

    block[0] = (x7 + x1) >> 8;
    block[1] = (x3 + x2) >> 8;
    block[2] = (x0 + x4) >> 8;
    block[3] = (x8 + x6) >> 8;
    block[4] = (x8 - x6) >> 8;
    block[5] = (x0 - x4) >> 8;
    block[6] = (x3 - x2) >> 8;
    block[7] = (x7 - x1) >> 8;
}

__device__ void idctCol(int* block) {
    int x0, x1, x2, x3, x4, x5, x6, x7;

    // Shortcut: if all AC terms are zero, directly scale the DC term
    if (!((x1 = (block[8*4]<<8)) | (x2 = block[8*6]) | (x3 = block[8*2]) | (x4 = block[8*1]) | (x5 = block[8*7]) | (x6 = block[8*5]) | (x7 = block[8*3]))) {
        block[8*0] = block[8*1] = block[8*2] = block[8*3] = block[8*4] = block[8*5] = block[8*6] = block[8*7] = clip((block[8*0]+32)>>6);
        return;
    }
    // Scale the DC coefficient
    x0 = (block[8*0]<<8) + 8192;

    int x8 = C7 * (x4 + x5) + 4;
    x4 = (x8 + (C1 - C7) * x4) >> 3;
    x5 = (x8 - (C1 + C7) * x5) >> 3;
    x8 = C3 * (x6 + x7) + 4;
    x6 = (x8 - (C3 - C5) * x6) >> 3;
    x7 = (x8 - (C3 + C5) * x7) >> 3;
    
    x8 = x0 + x1;
    x0 -= x1;
    x1 = C6 * (x3 + x2) + 4;
    x2 = (x1 - (C2 + C6) * x2) >> 3;
    x3 = (x1 + (C2 - C6) * x3) >> 3;
    x1 = x4 + x6;
    x4 -= x6;
    x6 = x5 + x7;
    x5 -= x7;

    x7 = x8 + x3;
    x8 -= x3;
    x3 = x0 + x2;
    x0 -= x2;
    x2 = (181 * (x4 + x5) + 128) >> 8;
    x4 = (181 * (x4 - x5) + 128) >> 8;

    block[8 * 0] = clip((x7 + x1) >> 14);
    block[8 * 1] = clip((x3 + x2) >> 14);
    block[8 * 2] = clip((x0 + x4) >> 14);
    block[8 * 3] = clip((x8 + x6) >> 14);
    block[8 * 4] = clip((x8 - x6) >> 14);
    block[8 * 5] = clip((x0 - x4) >> 14);
    block[8 * 6] = clip((x3 - x2) >> 14);
    block[8 * 7] = clip((x7 - x1) >> 14);
}

JPEGParser::JPEGParser(std::string& imagePath) {
    // Extract the file name of the image file from the file path
    fs::path file_path(imagePath);
    this->filename = file_path.filename().string();
    std::ifstream input(imagePath, std::ios::binary);
    
    std::vector<uint8_t> bytes((std::istreambuf_iterator<char>(input)), (std::istreambuf_iterator<char>()));
    this->readBytes = new uint8_t[bytes.size()];
    for (int i = 0; i < bytes.size(); i++) {
        readBytes[i] = bytes[i];
    }
    input.close();

    imageDataLength = 0;

    int zigzagEntries[64] = {
        0, 1, 5, 6, 14, 15, 27, 28,
        2, 4, 7, 13, 16, 26, 29, 42,
        3, 8, 12, 17, 25, 30, 41, 43,
        9, 11, 18, 24, 31, 40, 44, 53,
        10, 19, 23, 32, 39, 45, 52, 54,
        20, 22, 33, 38, 46, 51, 55, 60,
        21, 34, 37, 47, 50, 56, 59, 61,
        35, 36, 48, 49, 57, 58, 62, 63
    };

    hipMalloc((void**)&zigzag, 64 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(initialZigzag), zigzagEntries, sizeof(int) * 64);
}

/* Function to allocate the GPU space. */
void JPEGParser::move() {
    hipError_t err = hipMalloc((uint16_t**)&this->hf0codes, 256 * sizeof(uint16_t));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for hf0 codes: " << hipGetErrorString(err) << std::endl;
    }
    hipMalloc((uint16_t**)&this->hf0codes, 256 * sizeof(uint16_t));
    hipMalloc((uint16_t**)&this->hf1codes, 256 * sizeof(uint16_t));
    hipMalloc((uint16_t**)&this->hf16codes, 256 * sizeof(uint16_t));
    hipMalloc((uint16_t**)&this->hf17codes, 256 * sizeof(uint16_t));

    hipMalloc((int**)&this->hf0lengths, 256 * sizeof(int));
    hipMalloc((int**)&this->hf1lengths, 256 * sizeof(int));
    hipMalloc((int**)&this->hf16lengths, 256 * sizeof(int));
    hipMalloc((int**)&this->hf17lengths, 256 * sizeof(int));

    hipMemcpy(this->hf0codes, this->huffmanTrees[0]->codes, 256 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(this->hf1codes, this->huffmanTrees[1]->codes, 256 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(this->hf16codes, this->huffmanTrees[16]->codes, 256 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(this->hf17codes, this->huffmanTrees[17]->codes, 256 * sizeof(uint16_t), hipMemcpyHostToDevice);

    hipMemcpy(this->hf0lengths, this->huffmanTrees[0]->codeLengths, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(this->hf1lengths, this->huffmanTrees[1]->codeLengths, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(this->hf16lengths, this->huffmanTrees[16]->codeLengths, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(this->hf17lengths, this->huffmanTrees[17]->codeLengths, 256 * sizeof(int), hipMemcpyHostToDevice);
    
    // Allocating the channels in the GPU memory.
    hipMalloc((void**)&this->luminous, 64 * xBlocks * yBlocks * sizeof(int));
    hipMalloc((void**)&this->chromRed, 64 * xBlocks * yBlocks * sizeof(int));
    hipMalloc((void**)&this->chromYel, 64 * xBlocks * yBlocks * sizeof(int));
    hipMalloc((void**)&this->redOutput, 64 * xBlocks * yBlocks * sizeof(int));
    hipMalloc((void**)&this->greenOutput, 64 * xBlocks * yBlocks * sizeof(int));
    hipMalloc((void**)&this->blueOutput, 64 * xBlocks * yBlocks * sizeof(int));
}

void JPEGParser::extract() {        
    uint16_t tableSize = 0;
    uint8_t header = 0;
    this->quantTable1 = nullptr;
    this->quantTable2 = nullptr;
    this->imageData = nullptr;

    // Using the Stream class for reading bytes.
    Stream* stream = new Stream(this->readBytes);
    while (true) {
        uint16_t marker = stream->getMarker();

        if (marker == MARKERS[0]) {
            continue;
        } else if (marker == MARKERS[1]) {
            
            tableSize = stream->getMarker();
            this->applicationHeader = new uint8_t[(int) tableSize - 2];
            stream->getNBytes(this->applicationHeader, int(tableSize - 2));
            
        } else if (marker == MARKERS[2]) {
            
            stream->getMarker();
            uint8_t destination = stream->getByte();
            uint8_t* host_quantTable1 = new uint8_t[64];
            stream->getNBytes(host_quantTable1, 64);
            // hipMalloc((void**)&this->quantTable1, 64 * sizeof(uint8_t));
            hipError_t err = hipMalloc((void**)&this->quantTable1, 64 * sizeof(uint8_t));
            if (err != hipSuccess) {
                std::cerr << "CUDA malloc failed for quantTable1: " << hipGetErrorString(err) << std::endl;
            }
            err = hipMemcpy(this->quantTable1, host_quantTable1, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
            if (err != hipSuccess) {
                std::cerr << "CUDA memcpy failed for quantTable1: " << hipGetErrorString(err) << std::endl;
            }
            // hipMemcpy(this->quantTable1, host_quantTable1, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
            delete host_quantTable1;

            if(stream->getMarker() == MARKERS[2]) {
                stream->getMarker();
                destination = stream->getByte();
                this->quantTable2 = new uint8_t[64];
                uint8_t* host_quantTable2 = new uint8_t[64];
                stream->getNBytes(host_quantTable2, 64);
                // hipMalloc((void**)&this->quantTable2, 64 * sizeof(uint8_t));
                err = hipMalloc((void**)&this->quantTable2, 64 * sizeof(uint8_t));
                if (err != hipSuccess) {
                    std::cerr << "CUDA malloc failed for quantTable2: " << hipGetErrorString(err) << std::endl;
                }
                // hipMemcpy(this->quantTable2, host_quantTable2, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
                err = hipMemcpy(this->quantTable2, host_quantTable2, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
                if (err != hipSuccess) {
                    std::cerr << "CUDA memcpy failed for quantTable2: " << hipGetErrorString(err) << std::endl;
                }
                delete host_quantTable2;
                
            } else {
                std::cout << " Something went wrong at parsing second quant table." << std::endl;
            }
            
        } else if (marker == MARKERS[3]) {
            
            tableSize = stream->getMarker();
            this->startOfFrame = new uint8_t[(int) tableSize - 2];
            stream->getNBytes(this->startOfFrame, (int) tableSize - 2);
            Stream* frame = new Stream(this->startOfFrame);
            int precision = frame->getByte();
            this->height = frame->getMarker();
            this->width = frame->getMarker();
            this->paddedWidth = ((this->width + 7) / 8) * 8;
            this->paddedHeight = ((this->height + 7) / 8) * 8;
            this->xBlocks = this->paddedWidth / 8;
            this->yBlocks = this->paddedHeight / 8;
            delete frame;
            

        } else if (marker == MARKERS[4]) {
            
            tableSize = stream->getMarker();
            header = stream->getByte();
            this->huffmanTable1 = new uint8_t[(int) tableSize - 3];
            stream->getNBytes(this->huffmanTable1, (int) tableSize - 3);
            this->huffmanTrees[header] = new HuffmanTree(this->huffmanTable1);

            if (stream->getMarker() ==  MARKERS[4]) {
                tableSize = stream->getMarker();
                header = stream->getByte();
                this->huffmanTable2 = new uint8_t[(int) tableSize - 3];
                stream->getNBytes(this->huffmanTable2, (int) tableSize - 3);
                this->huffmanTrees[header] = new HuffmanTree(this->huffmanTable2);
            }

            if (stream->getMarker() ==  MARKERS[4]) {
                tableSize = stream->getMarker();
                header = stream->getByte();
                this->huffmanTable3 = new uint8_t[(int) tableSize - 3];
                stream->getNBytes(this->huffmanTable3, (int) tableSize - 3);
                this->huffmanTrees[header] = new HuffmanTree(this->huffmanTable3);
            }

            if (stream->getMarker() ==  MARKERS[4]) {
                tableSize = stream->getMarker();
                header = stream->getByte();
                this->huffmanTable4 = new uint8_t[(int) tableSize - 3];
                stream->getNBytes(this->huffmanTable4, (int) tableSize - 3);
                this->huffmanTrees[header] = new HuffmanTree(this->huffmanTable4);
            }
            
        } else if (marker == MARKERS[5]) {
            // std::cout <<"before" << std::endl;
            tableSize = stream->getMarker();
            this->startOfScan = new uint8_t[(int) tableSize - 2];
            stream->getNBytes(this->startOfScan, (int) tableSize - 2);
            uint8_t curByte, prevByte = 0x00;
            size_t size = 5 * 1024 * 1024;
            uint8_t* host_imageData = new uint8_t[size];
            
            while (true) {
                curByte = stream->getByte();
                if ((prevByte == 0xff) && (curByte == 0xd9))
                    break;
                if (curByte == 0x00) {
                    if (prevByte != 0xff) {
                        host_imageData[imageDataLength++] = curByte;
                    }
                } else {
                    host_imageData[imageDataLength++] = curByte;
                }
                prevByte = curByte;
            }
            
            imageDataLength--; // We remove the ending byte because it is extra 0xff.
            // hipMalloc((void**)&this->imageData, imageDataLength * sizeof(uint8_t));
            hipError_t err = hipMalloc((void**)&this->imageData, imageDataLength * sizeof(uint8_t));
            if (err != hipSuccess) {
                std::cerr << "CUDA malloc failed for imageData: " << hipGetErrorString(err) << std::endl;
            }
            err = hipMemcpy(this->imageData, host_imageData, imageDataLength * sizeof(uint8_t), hipMemcpyHostToDevice);
            if (err != hipSuccess) {
                std::cerr << "CUDA memcpy failed for imageData: " << hipGetErrorString(err) << std::endl;
            }
            // hipMemcpy(this->imageData, host_imageData, imageDataLength * sizeof(uint8_t), hipMemcpyHostToDevice);
            delete host_imageData;
            break;
        }
        // std::cout <<"after" << std::endl;
    } 
    
    delete stream;  
    move(); 
   
}

__device__ int match_huffman_code(uint8_t* stream, int bit_offset, uint16_t* huff_codes, int* huff_bits, int &code, int &length) {
    unsigned int extracted_bits = getNBits(stream, bit_offset, 16);
    // Compare against Huffman table
    for (int i = 0; i < 256; ++i) {
        if (huff_bits[i] > 0 && huff_bits[i] <= 16) { // Valid bit length
            unsigned int mask = (1 << huff_bits[i]) - 1;
            if ((extracted_bits >> (16 - huff_bits[i]) & mask) == huff_codes[i]) {
                code = i;
                length = huff_bits[i];
                return;
            }
        }
    }
}

__device__ int buildMCU(int* outBuffer, uint8_t* imageData, int bitOffset, uint8_t* quant, 
                        int& oldCoeff, uint16_t* dcHfcodes, int* dcHflengths, uint16_t* acHfcodes, int* acHflengths) {

    int code = 0;
    int code_length = 0;
    match_huffman_code(imageData, bitOffset, dcHfcodes, dcHflengths, code, code_length);
    bitOffset += code_length;
    uint16_t bits = getNBits(imageData, bitOffset, code);

    int decoded = decodeNumber(code, bits); 
    int dcCoeff = decoded + oldCoeff;
    //outBuffer[0] = dcCoeff * (int) quant[0];
    // printf("dc %d %d %d\n", dcCoeff, (int) quant[0], dcCoeff * (int) quant[0]);
    outBuffer[0] = dcCoeff;

    int length = 1;
    while (length < 64) {
        match_huffman_code(imageData, bitOffset, acHfcodes, acHflengths, code, code_length);
        bitOffset += code_length;
        if (code == 0) {
            break;
        }
        // The first part of the AC key length is the number of leading zeros
        if (code > 15) {
            length += (code >> 4);
            code = code & 0x0f;
        }
        bits = getNBits(imageData, bitOffset, code);
        if (length < 64) {
            decoded = decodeNumber(code, bits);
            int val;
            val = decoded * (int) quant[length];
            //outBuffer[length] = val;
            outBuffer[length] = decoded;
            // printf("ac %d %d %d\n", decoded, (int) quant[length], decoded * (int) quant[length]);
            length++;
        }
    }
    // Update oldCoeff for the next MCU
    oldCoeff = dcCoeff;
    return bitOffset;
}

JPEGParser::~JPEGParser() {
    // std::cout << "destructor1" << std::endl;
    if (idctTable) hipFree(idctTable);
    // std::cout << "destructor2" << std::endl;
    // if (channels) delete channels;
    // std::cout << "destructor3" << std::endl;
    for (auto& tree : huffmanTrees) {
       if (tree.second) delete tree.second;
    }
    // std::cout << "destructor4" << std::endl;
    if (quantTable1) hipFree(this->quantTable1);
    // std::cout << "destructor5" << std::endl;
    if (quantTable2) hipFree(this->quantTable2);
    // std::cout << "destructor6" << std::endl;
    if (imageData) hipFree(this->imageData);
    // std::cout << "destructor7" << std::endl;
    if (readBytes) delete[] this->readBytes;
    // std::cout << "destructor8" << std::endl;
    if (applicationHeader) delete[] this->applicationHeader;
    // std::cout << "destructor9" << std::endl;
    if (startOfFrame) delete[] this->startOfFrame;
    // std::cout << "destructor10" << std::endl;
    if (startOfScan) delete[] this->startOfScan;
    // std::cout << "destructor11" << std::endl;
    if (huffmanTable1) delete[] this->huffmanTable1;
    // std::cout << "destructor12" << std::endl;
    if (huffmanTable2) delete[] this->huffmanTable2;
    // std::cout << "destructor13" << std::endl;
    if (huffmanTable3) delete[] this->huffmanTable3;
    // std::cout << "destructor14" << std::endl;
    if (huffmanTable4) delete[] this->huffmanTable4;
    // std::cout << "destructor15" << std::endl;
}

__device__ void performHuffmanDecoding(uint8_t* imageData, int* arr_l, int* arr_r, int* arr_y,
                                       uint8_t* quant1, uint8_t* quant2,
                                       uint16_t* hf0codes, int* hf0lengths, uint16_t* hf16codes, int* hf16lengths,
                                       uint16_t* hf1codes, int* hf1lengths, uint16_t* hf17codes, int* hf17lengths,
                                       int yBlocks, int xBlocks) {
    int* curLuminous = arr_l;
    int* curChromRed = arr_r;
    int* curChromYel = arr_y;
    int oldLumCoeff = 0, oldCbdCoeff = 0, oldCrdCoeff = 0;
    int bitOffset = 0;

    for (int y = 0; y < yBlocks; y++) {
        for (int x = 0; x < xBlocks; x++) {
            bitOffset = buildMCU(curLuminous, imageData, bitOffset, quant1, oldLumCoeff, hf0codes, hf0lengths, hf16codes, hf16lengths);
            bitOffset = buildMCU(curChromRed, imageData, bitOffset, quant2, oldCbdCoeff, hf1codes, hf1lengths, hf17codes, hf17lengths);
            bitOffset = buildMCU(curChromYel, imageData, bitOffset, quant2, oldCrdCoeff, hf1codes, hf1lengths, hf17codes, hf17lengths);
            curLuminous += 64;
            curChromRed += 64;
            curChromYel += 64;
        }
    }
}

__device__ void performZigzagReordering(int* arr_l, int* arr_r, int* arr_y, 
                                        int* zigzag_l, int* zigzag_r, int* zigzag_y,
                                        int blockIndex, int threadIndexInBlock, int threadId,
                                        const int* initialZigzag) {
    zigzag_l[threadId] = arr_l[blockIndex * 64 + initialZigzag[threadIndexInBlock]];
    zigzag_r[threadId] = arr_r[blockIndex * 64 + initialZigzag[threadIndexInBlock]];
    zigzag_y[threadId] = arr_y[blockIndex * 64 + initialZigzag[threadIndexInBlock]];
}

__device__ void performColorConversion(int* arr_l, int* arr_r, int* arr_y,
                                       int* redOutput, int* greenOutput, int* blueOutput,
                                       int totalPixels, int width, int threadId, int blockDimGridDim) {
    for (int i = threadId; i < totalPixels; i += blockDimGridDim) {
        int blockId = i / 64;
        int blockRow = blockId / (width / 8);
        int blockColumn = blockId % (width / 8);

        int rowStart = blockRow * 8;
        int columnStart = blockColumn * 8;

        int pixelIndexInBlock = i % 64;
        int rowInBlock = pixelIndexInBlock / 8;
        int columnInBlock = pixelIndexInBlock % 8;

        int globalRow = rowStart + rowInBlock;
        int globalColumn = columnStart + columnInBlock;

        int actualIndex = globalRow * width + globalColumn;

        // Retrieve pixel data and perform the color conversion
        float red = arr_y[i] * (2 - 2 * 0.299) + arr_l[i];
        float blue = arr_r[i] * (2 - 2 * 0.114) + arr_l[i];
        float green = (arr_l[i] - 0.114 * blue - 0.299 * red) / 0.587;

        // Clamp values to [0, 255]
        redOutput[actualIndex] = min(max(static_cast<int>(red + 128), 0), 255);
        greenOutput[actualIndex] = min(max(static_cast<int>(green + 128), 0), 255);
        blueOutput[actualIndex] = min(max(static_cast<int>(blue + 128), 0), 255);
    }
}

__global__ void decodeKernel(uint8_t* imageData, int* arr_l, int* arr_r, int* arr_y, double* idctTable, int validHeight, 
                                int validWidth, int width, int height, int xBlocks, int yBlocks, int* redOutput, 
                                int* greenOutput, int* blueOutput, uint8_t* quant1, uint8_t* quant2, 
                                uint16_t* hf0codes, uint16_t* hf1codes, uint16_t* hf16codes, uint16_t* hf17codes,
                                int* hf0lengths, int* hf1lengths, int* hf16lengths, int* hf17lengths) {

    // Thread and block IDs
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    // Serial section - only one thread in one block
    if (blockX == 0 && blockY == 0 && threadX == 0 && threadY == 0) {
        performHuffmanDecoding(imageData, arr_l, arr_r, arr_y, quant1, quant2, 
                               hf0codes, hf0lengths, hf16codes, hf16lengths, 
                                hf1codes, hf1lengths, hf17codes, hf17lengths, yBlocks, xBlocks);
        global_sync_flag = 1; // Mark the serial work as complete
    }


    // Ensure all blocks wait until the serial work is done
    if (threadX == 0 && threadY == 0) {
        while (atomicAdd(&global_sync_flag, 0) == 0) {
            // Spin until the serial section is complete
        }
    }
    __syncthreads(); // Synchronize all threads within the block
    


   // Shared memory for zigzag arrays
    __shared__ int sharedZigzag[3 * 64];
    int* zigzag_l = &sharedZigzag[0];
    int* zigzag_r = &sharedZigzag[64];
    int* zigzag_y = &sharedZigzag[128];

    int globalBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
    int blockStart = globalBlockIndex * 64;
    

    // Identify the thread's position in the 8x8 grid
    int threadRow = threadIdx.y; // Row index (0-7)
    int threadCol = threadIdx.x; // Column index (0-7)
    int threadIndexInBlock = threadRow * 8 + threadCol; // Flattened index

    // Calculate the global index for this thread
    int globalIndex = blockStart + threadIndexInBlock;

    // zigzag_l[threadIndexInBlock] = arr_l[blockStart + initialZigzag[threadIndexInBlock]];
    // zigzag_r[threadIndexInBlock] = arr_r[blockStart + initialZigzag[threadIndexInBlock]];
    // zigzag_y[threadIndexInBlock] = arr_y[blockStart + initialZigzag[threadIndexInBlock]];

    zigzag_l[threadIndexInBlock] = arr_l[blockStart + initialZigzag[threadIndexInBlock]] * (int) quant1[initialZigzag[threadIndexInBlock]];
    zigzag_r[threadIndexInBlock] = arr_r[blockStart + initialZigzag[threadIndexInBlock]] * (int) quant2[initialZigzag[threadIndexInBlock]];
    zigzag_y[threadIndexInBlock] = arr_y[blockStart + initialZigzag[threadIndexInBlock]] * (int) quant2[initialZigzag[threadIndexInBlock]];

    __syncthreads();

    // if (threadIndexInBlock == 0 && globalBlockIndex < 192) {
    //     printf("%d %d %d \n", zigzag_l[threadIndexInBlock], (int) quant1[threadIndexInBlock], zigzag_l[threadIndexInBlock]*(int) quant1[threadIndexInBlock]);
    //     // counter++;
    // }

    if (threadCol == 0) {
        idctRow(zigzag_l + threadIndexInBlock);
        idctRow(zigzag_r + threadIndexInBlock);
        idctRow(zigzag_y + threadIndexInBlock);
    }

    __syncthreads();


    if (threadRow == 0) {
        idctCol(zigzag_l + threadIndexInBlock);
        idctCol(zigzag_r + threadIndexInBlock);
        idctCol(zigzag_y + threadIndexInBlock);
        
    }

    __syncthreads();
    arr_l[globalIndex] = zigzag_l[threadIndexInBlock];
    arr_r[globalIndex] = zigzag_r[threadIndexInBlock];
    arr_y[globalIndex] = zigzag_y[threadIndexInBlock];
    // if (threadCol < validWidth && threadRow < validHeight) {
    //     double localSum_l = 0.0;
    //     double localSum_r = 0.0;
    //     double localSum_y = 0.0;
    //     for (int u = 0; u < 8; u++) {
    //         for (int v = 0; v < 8; v++) {
    //             localSum_l += zigzag_l[v * 8 + u] * idctTable[u * 8 + threadCol] * idctTable[v * 8 + threadRow];
    //             localSum_r += zigzag_r[v * 8 + u] * idctTable[u * 8 + threadCol] * idctTable[v * 8 + threadRow];
    //             localSum_y += zigzag_y[v * 8 + u] * idctTable[u * 8 + threadCol] * idctTable[v * 8 + threadRow];
    //         }
    //     }

    //     arr_l[globalIndex] = static_cast<int>(std::floor(localSum_l / 4.0)); //luminuous
    //     arr_y[globalIndex] = static_cast<int>(std::floor(localSum_y / 4.0)); //chromyel
    //     arr_r[globalIndex] = static_cast<int>(std::floor(localSum_r / 4.0)); // chromred
    // }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x; // x-coordinate
    int y = blockIdx.y * blockDim.y + threadIdx.y; // y-coordinate
    int i = y * width + x;

    if (x < width && y < height) {
        int blockIndex = (y / 8) * xBlocks + (x / 8); // Index of the current 8x8 block
        int pixelIndexInBlock = threadIdx.y * 8 + threadIdx.x;  // Position within the block

        float red = arr_y[blockIndex * 64 + pixelIndexInBlock] * (2 - 2 * 0.299) + arr_l[blockIndex * 64 + pixelIndexInBlock];
        float blue = arr_r[blockIndex * 64 + pixelIndexInBlock] * (2 - 2 * 0.114) + arr_l[blockIndex * 64 + pixelIndexInBlock];
        float green = (arr_l[blockIndex * 64 + pixelIndexInBlock] - 0.114 * blue - 0.299 * red) / 0.587;

        int castedRed = static_cast<int>(red + 128);
        int castedGreen = static_cast<int>(green + 128);
        int castedBlue = static_cast<int>(blue + 128);

        if (castedRed > 255) {
            redOutput[i] = 255;
        } else if (castedRed < 0) {
            redOutput[i] = 0;
        } else {
            redOutput[i] = castedRed;
        }

        if (castedGreen > 255) {
            greenOutput[i] = 255;
        } else if (castedGreen < 0) {
            greenOutput[i] = 0;
        } else {
            greenOutput[i] = castedGreen;
        }

        if (castedBlue > 255) {
            blueOutput[i] = 255;
        } else if (castedBlue < 0) {
            blueOutput[i] = 0;
        } else {
            blueOutput[i] = castedBlue;
        }
    }
}

void JPEGParser::decode() {
    dim3 blockSize(8, 8);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    // size_t channelSize = width * height * sizeof(int);

    decodeKernel<<<gridSize, blockSize>>>(this->imageData, this->luminous, this->chromRed, this->chromYel, idctTable, 8, 8,  
                                            this->width, this->height, this->xBlocks, this->yBlocks, this->redOutput, this->greenOutput, this->blueOutput,
                                            this->quantTable1, this->quantTable2, this->hf0codes, this->hf1codes, this->hf16codes, this->hf17codes, 
                                            this->hf0lengths, this->hf1lengths, this->hf16lengths, this->hf17lengths);

    

    if (luminous) hipFree(luminous);
    if (chromRed) hipFree(chromRed);
    if (chromYel) hipFree(chromYel);
    if (hf0codes) hipFree(hf0codes);
    if (hf1codes) hipFree(hf1codes);
    if (hf16codes) hipFree(hf16codes);
    if (hf17codes) hipFree(hf17codes);
    if (hf0lengths) hipFree(hf0lengths);
    if (hf1lengths) hipFree(hf1lengths);
    if (hf16lengths) hipFree(hf16lengths);
    if (hf17lengths) hipFree(hf17lengths); 
}

void JPEGParser::write() {
    this->channels = new ImageChannels(this->height * this->width);
    size_t channelSize = this->width * this->height * sizeof(int);
    hipMemcpy(channels->getR().data(), redOutput, channelSize, hipMemcpyDeviceToHost);
    hipMemcpy(channels->getG().data(), greenOutput, channelSize, hipMemcpyDeviceToHost);
    hipMemcpy(channels->getB().data(), blueOutput, channelSize, hipMemcpyDeviceToHost);
    hipFree(redOutput);
    hipFree(greenOutput);
    hipFree(blueOutput);

    // Writing the decoded channels to a file instead of displaying using opencv
    fs::path output_dir = "../testing/cudaUF_output_arrays"; // Change the directory name here for future CUDA implementations
    fs::path full_path = output_dir / this->filename;
    full_path.replace_extension(".array");
    std::ofstream outfile(full_path);
    outfile << this->height << " " << this->width << std::endl;
    std::copy(this->channels->getR().begin(), this->channels->getR().end(), std::ostream_iterator<int>(outfile, " "));
    outfile << std::endl;
    std::copy(this->channels->getG().begin(), this->channels->getG().end(), std::ostream_iterator<int>(outfile, " "));
    outfile << std::endl;
    std::copy(this->channels->getB().begin(), this->channels->getB().end(), std::ostream_iterator<int>(outfile, " "));
    outfile.close();
    delete channels;
}