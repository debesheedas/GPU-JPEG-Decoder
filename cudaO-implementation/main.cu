#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include "src/parser.h"

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "Please provide the name of the image file to be decompressed." << std::endl;
        return 1;
    }

    // Reading the image bytes
    std::string imagePath = argv[1];
    // Extract the file name of the image file from the file path
    fs::path file_path(imagePath);
    std::string filename = file_path.filename().string();

    uint16_t* hfCodes; 
    int* hfLengths;
    uint8_t* quantTables;
    int16_t* yCrCbChannels;
    int16_t* rgbChannels;
    int16_t* outputChannels;
    int* zigzagLocations;

    uint8_t* imageData;
    int width = 0;
    int height = 0;
    std::unordered_map<int,HuffmanTree*> huffmanTrees;

    // Extracting the byte chunks
    extract(imagePath, quantTables, imageData, width, height, huffmanTrees);
    // Allocating memory for the arrays
    allocate(hfCodes, hfLengths, huffmanTrees, yCrCbChannels, rgbChannels, outputChannels, width, height, zigzagLocations);
    
    decodeKernel<<<1, 256>>>(imageData, yCrCbChannels, rgbChannels, outputChannels, width, height, quantTables, hfCodes, hfLengths, zigzagLocations);
    hipDeviceSynchronize();
    
    write(outputChannels, width, height, filename);
    clean(hfCodes, hfLengths, quantTables, yCrCbChannels, rgbChannels, outputChannels, zigzagLocations, imageData, huffmanTrees);
}