#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include "src/parser.h"
#include <nvtx3/nvToolsExt.h>

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "Please provide the name of the image file to be decompressed." << std::endl;
        return 1;
    }

    std::string imagePath = argv[1];
    fs::path file_path(imagePath);
    std::string filename = file_path.filename().string();

    uint16_t* hfCodes; 
    int* hfLengths;
    uint8_t* quantTables;
    int16_t* yCrCbChannels;
    int16_t* rgbChannels;
    int16_t* outputChannels;
    int* zigzagLocations;

    uint8_t* imageData;
    int imageDataLength;
    int* sInfo;
    int width = 0;
    int height = 0;
    std::unordered_map<int, HuffmanTree*> huffmanTrees;

    extract(imagePath, quantTables, imageData, imageDataLength, width, height, huffmanTrees);
    allocate(hfCodes, hfLengths, huffmanTrees, yCrCbChannels, rgbChannels, outputChannels, width, height, zigzagLocations, sInfo, 1024);
    
    decodeKernel<<<1, 1024>>>(imageData, imageDataLength, yCrCbChannels, rgbChannels, outputChannels, width, height, quantTables, hfCodes, hfLengths, zigzagLocations, sInfo);
    hipDeviceSynchronize();

    write(outputChannels, width, height, filename);
    clean(hfCodes, hfLengths, quantTables, yCrCbChannels, rgbChannels, outputChannels, zigzagLocations, imageData, huffmanTrees, sInfo);
}