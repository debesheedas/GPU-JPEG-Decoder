#include "hip/hip_runtime.h"
#include "parser.h"

__global__ void initializeIDCTTableKernel(double *dIdctTable, int numThreads)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < numThreads) {
        double normCoeff = ((id / 8) == 0) ? (1.0 / sqrt(2.0)) : 1.0;
        dIdctTable[id] = normCoeff * cos(((2.0 * (id%8) + 1.0) * (id/8) * M_PI) / 16.0);
    }
}

JPEGParser::JPEGParser(std::string& imagePath): quantTables(2) {
    // Extract the file name of the image file from the file path
    fs::path file_path(imagePath);
    this->filename = file_path.filename().string();
    std::ifstream input(imagePath, std::ios::binary);
    
    std::vector<uint8_t> bytes((std::istreambuf_iterator<char>(input)), (std::istreambuf_iterator<char>()));
    this->readBytes = bytes;
    input.close();
    
    quantTables[0] = new uint8_t[64];
    quantTables[1] = new uint8_t[64];

    int blockSize = 64;
    int gridSize = (64 + blockSize - 1) / blockSize;
    hipMalloc((void**)&idctTable, 64 * sizeof(double));
    initializeIDCTTableKernel<<<blockSize, gridSize>>>(idctTable, 64);

    //hipMalloc((void**)&quantTables[0], 64 * sizeof(uint8_t));
    //hipMalloc((void**)&quantTables[1], 64 * sizeof(uint8_t));
    // JPEGParser::extract(bytes);
}

void JPEGParser::extract() {        
    uint16_t tableSize = 0;
    uint8_t header = 0;

    // Using the Stream class for reading bytes.
    Stream* stream = new Stream(this->readBytes);

    while (true) {
        uint16_t marker = stream->getMarker();

        if (marker == MARKERS[0]) {
            continue;
        } else if (marker == MARKERS[1]) {
            // std::cout<< "Extracting Application Header" << std::endl;
            tableSize = stream->getMarker();
            stream->getNBytes(this->applicationHeader, int(tableSize - 2));
        } else if (marker == MARKERS[2]) {
            // std::cout<< "Extracting Quant Tables" << std::endl;
            stream->getMarker();
            uint8_t destination = stream->getByte();
            stream->getNBytes(quantTables[0], 64);
            //std::cout << " got the goods **** " << std::endl;
            ///hipMemcpy(quantTables[0], temp, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
            if(stream->getMarker() == MARKERS[2]) {
                stream->getMarker();
                destination = stream->getByte();
                stream->getNBytes(quantTables[1], 64);
                //std::cout << " got the goods " << std::endl;
                //hipMemcpy(quantTables[1], temp, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
            } else {
                std::cout << " Something went wrong at parsing second quant table." << std::endl;
            }
        } else if (marker == MARKERS[3]) {
            // std::cout<< "Extracting Start of Frame" << std::endl;
            tableSize = stream->getMarker();
            stream->getNBytes(this->startOfFrame, (int) tableSize - 2);
            Stream* frame = new Stream(this->startOfFrame);
            int precision = frame->getByte();
            this->height = frame->getMarker();
            this->width = frame->getMarker();
        } else if (marker == MARKERS[4]) {
            // std::cout<< "Extracting Huffman Tables" << std::endl;
            tableSize = stream->getMarker();
            header = stream->getByte();
            stream->getNBytes(this->huffmanTables[0], (int) tableSize - 3);
            this->huffmanTrees[header] = new HuffmanTree(this->huffmanTables[0]);

            int huffmanCount = 1;
            while(huffmanCount < 4) {
                if (stream->getMarker() ==  MARKERS[4]) {
                    tableSize = stream->getMarker();
                    header = stream->getByte();
                    stream->getNBytes(this->huffmanTables[huffmanCount], (int) tableSize - 3);
                    this->huffmanTrees[header] = new HuffmanTree(this->huffmanTables[huffmanCount]);
                    huffmanCount++; 
                }
            }
        } else if (marker == MARKERS[5]) {
            // std::cout<< "Start of Scan" << std::endl;
            tableSize = stream->getMarker();
            stream->getNBytes(this->startOfScan, (int) tableSize - 2);
            uint8_t curByte, prevByte = 0x00;

            while (true) {
                curByte = stream->getByte();
                if ((prevByte == 0xff) && (curByte == 0xd9))
                    break;
                if (curByte == 0x00) {
                    if (prevByte != 0xff) {
                        this->imageData.push_back(curByte);
                    }
                } else {
                    this->imageData.push_back(curByte);
                }
                prevByte = curByte;
            }
            
            imageData.pop_back(); // We remove the ending byte because it is extra 0xff.
            break;
        }
    }   
}

void JPEGParser::buildMCU(int* arr, Stream* imageStream, int hf, int quant, int& oldCoeff, int validWidth = 8, int validHeight = 8) {
    std::vector<int> hostBuffer(64,0);
    uint8_t code = this->huffmanTrees[hf]->getCode(imageStream);
    uint16_t bits = imageStream->getNBits(code);
    int decoded = Stream::decodeNumber(code, bits);
    int dcCoeff = decoded + oldCoeff;

    hostBuffer[0] = dcCoeff * (int) this->quantTables[quant][0];
    int length = 1;

    while (length < 64) {
        code = this->huffmanTrees[16 + hf]->getCode(imageStream);

        if (code == 0) {
            break;
        }

        // The first part of the AC key length is the number of leading zeros
        if (code > 15) {
            length += (code >> 4);
            code = code & 0x0f;
        }

        bits = imageStream->getNBits(code);
        if (length < 64) {
            decoded = Stream::decodeNumber(code, bits);
            int val = decoded * (int) this->quantTables[quant][length];
            hostBuffer[length] = val;
            length++;
        }
    }

    // Create and process the IDCT for this block with the valid dimensions
    hipMemcpy(arr, hostBuffer.data(), 64*sizeof(int), hipMemcpyHostToDevice);
    IDCT* idct = new IDCT(arr, idctTable);
    idct->rearrangeUsingZigzag(validWidth, validHeight);
    idct->performIDCT(validWidth, validHeight);

    // Update oldCoeff for the next MCU
    oldCoeff = dcCoeff;

    delete idct;
}


void JPEGParser::decode() {
    int oldLumCoeff = 0;
    int oldCbdCoeff = 0;
    int oldCrdCoeff = 0;

    // Pad the image dimension if it is not divisible by 8
    int paddedWidth = ((this->width + 7) / 8) * 8;
    int paddedHeight = ((this->height + 7) / 8) * 8;

    int xBlocks = paddedWidth / 8;
    int yBlocks = paddedHeight / 8;

    Stream* imageStream = new Stream(this->imageData);
    std::vector<std::vector<std::vector<int>>> luminous(xBlocks, std::vector<std::vector<int>>(yBlocks, std::vector<int>(64,0)));
    std::vector<std::vector<std::vector<int>>> chromRed(xBlocks, std::vector<std::vector<int>>(yBlocks, std::vector<int>(64,0)));
    std::vector<std::vector<std::vector<int>>> chromYel(xBlocks, std::vector<std::vector<int>>(yBlocks, std::vector<int>(64,0)));

    int* temp;
    hipMalloc((void**)&temp, 64 * sizeof(int));

    for (int y = 0; y < yBlocks; y++) {
        for (int x = 0; x < xBlocks; x++) {
            // Determine the valid width and height for this block to account for padding
            int blockWidth = (x == xBlocks - 1 && paddedWidth != this->width) ? this->width % 8 : 8;
            int blockHeight = (y == yBlocks - 1 && paddedHeight != this->height) ? this->height % 8 : 8;

            this->buildMCU(temp, imageStream, 0, 0, oldLumCoeff, blockWidth, blockHeight);
            hipMemcpy(luminous[x][y].data(), temp, 64 * sizeof(int), hipMemcpyDeviceToHost);
            this->buildMCU(temp, imageStream, 1, 1, oldCbdCoeff, blockWidth, blockHeight);
            hipMemcpy(chromRed[x][y].data(), temp, 64 * sizeof(int), hipMemcpyDeviceToHost);
            this->buildMCU(temp, imageStream, 1, 1, oldCrdCoeff, blockWidth, blockHeight);
            hipMemcpy(chromYel[x][y].data(), temp, 64 * sizeof(int), hipMemcpyDeviceToHost);
        }
    }

    this->channels = new ImageChannels(this->height * this->width);

    // Write the processed data into the channels, ignoring padded regions
    for (int y = 0; y < yBlocks; y++) {
        for (int x = 0; x < xBlocks; x++) {
            for (int i = 0; i < 8; i++) {
                for (int j = 0; j < 8; j++) {
                    int pixelY = y * 8 + i;
                    int pixelX = x * 8 + j;

                    if (pixelY < this->height && pixelX < this->width) {
                        int index = i * 8 + j;
                        int pixelIndex = pixelY * this->width + pixelX;

                        this->channels->getY()[pixelIndex] = luminous[x][y][index];
                        this->channels->getCr()[pixelIndex] = chromYel[x][y][index];
                        this->channels->getCb()[pixelIndex] = chromRed[x][y][index];
                    }
                }
            }
        }
    }

    // Convert YCbCr channels to RGB
    colorConversion(this->channels->getY(), this->channels->getCr(), this->channels->getCb(), this->channels->getR(), this->channels->getG(), this->channels->getB(), this->height * this->width);

}

void JPEGParser::write() {
    // Writing the decoded channels to a file instead of displaying using opencv
    fs::path output_dir = "../testing/cuda1_output_arrays"; // Change the directory name here for future CUDA implementations
    fs::path full_path = output_dir / this->filename;
    full_path.replace_extension(".array");
    std::ofstream outfile(full_path);
    outfile << this->height << " " << this->width << std::endl;
    std::copy(this->channels->getR().begin(), this->channels->getR().end(), std::ostream_iterator<int>(outfile, " "));
    outfile << std::endl;
    std::copy(this->channels->getG().begin(), this->channels->getG().end(), std::ostream_iterator<int>(outfile, " "));
    outfile << std::endl;
    std::copy(this->channels->getB().begin(), this->channels->getB().end(), std::ostream_iterator<int>(outfile, " "));
    outfile.close();
}