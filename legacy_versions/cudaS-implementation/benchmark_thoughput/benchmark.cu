#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <string>
#include <vector>
#include <chrono>
#include <fstream>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include "/home/dphpc2024_jpeg_1/GPU-JPEG-Decoder/cudaS-implementation/src/parser.h"

namespace fs = std::filesystem;

// Function to get all images in the dataset
std::vector<std::string> getAllImages(const std::string& datasetPath) {
    std::vector<std::string> imagePaths;
    for (const auto& entry : fs::recursive_directory_iterator(datasetPath)) {
        if (entry.is_regular_file() && entry.path().extension() == ".jpeg") {
            imagePaths.push_back(entry.path().string());
        }
    }
    return imagePaths;
}
// __global__ void myKernel(int size) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < size) {
//         idx = idx; // Example operation
//     }
// }
// CUDA kernel for parallel image processing (dummy example, replace with actual implementation)


// void copyToDevice(HostData* parser, DeviceData* data) {
//     data->imageData = parser->imageData;
//     data->yCrCbChannels = parser->yCrCbChannels;
//     data->outputChannels = parser->outputChannels;
//     data->zigzagLocations = parser->zigzagLocations;
//     data->width = parser->width;
//     data->height = parser->height;
//     data->rgbChannels = parser->rgbChannels;
//     data->quantTables = parser->quantTables;
//     data->hfCodes = parser->hfCodes;
//     data->hfLengths = parser->hfLengths;
// }

// Benchmark function for throughput measurement
void JPEGDecoderBenchmark(benchmark::State& state, std::vector<std::string> imagePaths) {
    size_t numImages = imagePaths.size();
    std::ofstream resultFile("benchmark_results.txt", std::ios_base::app);

    // Define batch size (adjust based on available memory)
    size_t batchSize = 1000; // Example batch size
    int threads = 32;
    size_t numBatches = (numImages + batchSize - 1) / batchSize;
    std::cout<< "num batches " << numBatches << " | numImages " << numImages << std::endl;
    
    for (auto _ : state) {
        float totalKernelTime = 0.0f; // Total time across all batches
        
        DeviceData structs[batchSize];
        DeviceData* deviceStructs;
        hipMalloc(&deviceStructs, batchSize * sizeof(DeviceData));
        // std::cout<<"Allocate Complete"<<std::endl;
        
        for (size_t batchIdx = 0; batchIdx < numBatches; ++batchIdx) {
            HostData hosts[batchSize];
            size_t startIdx = batchIdx * batchSize;
            size_t endIdx = std::min(startIdx + batchSize, numImages);
            size_t currentBatchSize = endIdx - startIdx;

            for (size_t i = 0; i < currentBatchSize; ++i) {
                
                size_t globalIdx = startIdx + i;
                HostData* host_data = &hosts[i];
                DeviceData* data = &structs[i];
        
                host_data->imagePath = imagePaths[globalIdx];
                extract(host_data->imagePath, data->quantTables, data->imageData, data->width, data->height, host_data->huffmanTrees);
                allocate(data->hfCodes, data->hfLengths, host_data->huffmanTrees, data->yCrCbChannels, data->rgbChannels, data->outputChannels, data->width, data->height, data->zigzagLocations);
            }
            // Allocate memory for the current batch on the GPU
            hipMemcpy(deviceStructs, structs, currentBatchSize * sizeof(DeviceData), hipMemcpyHostToDevice);
            hipEvent_t batchStart, batchStop;
            hipEventCreate(&batchStart);
            hipEventCreate(&batchStop);

            //nvtxRangePushA("BatchDecodeKernel Execution");
            hipEventRecord(batchStart);
            batchDecodeKernel<<<currentBatchSize, threads>>>(deviceStructs);
            hipEventRecord(batchStop);
            //nvtxRangePop();  // End NVTX marker
            
            hipEventSynchronize(batchStop);
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(err));
            }
            // Calculate time for this batch
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, batchStart, batchStop);
            totalKernelTime += milliseconds;
            // Temporarily write output
            // for (size_t i = 0; i < currentBatchSize; ++i) {
            //     // size_t globalIdx = startIdx + i;
            //     // std::cout << "debug2" <<std::endl;
            //     jpegParsers[i]->write();
            // }
            hipEventDestroy(batchStart);
            hipEventDestroy(batchStop);
            // Cleanup for this batch
            for (size_t i = 0; i < currentBatchSize; ++i) {
                HostData* host_data = &hosts[i];
                DeviceData* data = &structs[i];
                // std::cout << host_data->huffmanTrees[0]->codes[0] << std::endl;
                clean(data->hfCodes, data->hfLengths, data->quantTables, data->yCrCbChannels, data->rgbChannels, data->outputChannels, data->zigzagLocations, data->imageData, host_data->huffmanTrees);
            }
        }
        double seconds = totalKernelTime / 1000.0;
        // Calculate throughput
        double throughput = numImages / seconds; // Images per second
        double totalBytesProcessed = 0.0;
        for (const auto& path : imagePaths) {
            totalBytesProcessed += fs::file_size(path);  // Calculate total bytes processed
        }
        double bytesPerSecond = totalBytesProcessed / seconds; // bytes per second
        // Set iteration metrics
        state.SetIterationTime(seconds);
        state.counters["throughput_images_per_sec"] = throughput;
        state.counters["bytes_per_sec"] = bytesPerSecond;
        if (deviceStructs) hipFree(deviceStructs);

        // Log results
        resultFile << "Throughput: " << throughput << " images/sec, "
                   << "Bytes per second: " << bytesPerSecond / (1024 * 1024) << " MB/sec\n";
    }
    
    resultFile.close();
}

int main(int argc, char** argv) {
    std::string datasetPath = "/home/dphpc2024_jpeg_1/GPU-JPEG-Decoder/benchmarking_dataset_through";

    std::vector<std::string> imagePaths = getAllImages(datasetPath);

    if (imagePaths.empty()) {
        std::cout << "No images found in the dataset directory." << std::endl;
        return 1;
    }

    benchmark::RegisterBenchmark("BM_JPEGDecoder_Throughput", [imagePaths](benchmark::State& state) {
        JPEGDecoderBenchmark(state, imagePaths);
    })
    ->Unit(benchmark::kMillisecond)
    ->Iterations(10);

    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}
