#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <string>
#include <vector>
#include <chrono>
#include <fstream>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include "/home/dphpc2024_jpeg_1/GPU-JPEG-Decoder/cudaU-implementation/src/parser.h"

namespace fs = std::filesystem;

// Function to get all images in the dataset
std::vector<std::string> getAllImages(const std::string& datasetPath) {
    std::vector<std::string> imagePaths;
    for (const auto& entry : fs::recursive_directory_iterator(datasetPath)) {
        if (entry.is_regular_file() && entry.path().extension() == ".jpeg") {
            imagePaths.push_back(entry.path().string());
        }
    }
    return imagePaths;
}

// __global__ void myKernel(int size) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < size) {
//         idx = idx; // Example operation
//     }
// }
// CUDA kernel for parallel image processing (dummy example, replace with actual implementation)

struct JPEGParserData {
    unsigned char* imageData;   // Pointer to image data
    int* luminous;            // Pointer to luminance data
    int* chromRed;            // Pointer to chroma red data
    int* chromYel;            // Pointer to chroma yellow data            // Pointer to chroma yellow data
    double* idctTable;           // Pointer to IDCT table
    int idctWidth;              // Width of IDCT (e.g., 8)
    int idctHeight;             // Height of IDCT (e.g., 8)
    int width;                  // Image width
    int height;                 // Image height
    int xBlocks;                // Number of horizontal blocks
    int yBlocks;                // Number of vertical blocks
    int* redOutput;           // Pointer to red channel output
    int* greenOutput;         // Pointer to green channel output
    int* blueOutput;          // Pointer to blue channel output
    uint8_t* quantTable1;         // Pointer to first quantization table
    uint8_t* quantTable2;         // Pointer to second quantization table
    uint16_t* hf0codes;    // Huffman table 0 codes
    uint16_t* hf1codes;    // Huffman table 1 codes
    uint16_t* hf16codes;   // Huffman table 16 codes
    uint16_t* hf17codes;   // Huffman table 17 codes
    int* hf0lengths;            // Huffman table 0 lengths
    int* hf1lengths;            // Huffman table 1 lengths
    int* hf16lengths;           // Huffman table 16 lengths
    int* hf17lengths;           // Huffman table 17 lengths
};

JPEGParserData copyToStruct(JPEGParser* parser) {
    JPEGParserData data;

    // Copy the pointers and scalar values
    data.imageData = parser->imageData;
    data.luminous = parser->luminous;
    data.chromRed = parser->chromRed;
    data.chromYel = parser->chromYel;
    data.idctTable = parser->idctTable;
    data.idctWidth = 8;  // Fixed width
    data.idctHeight = 8; // Fixed height
    data.width = parser->width;
    data.height = parser->height;
    data.xBlocks = parser->xBlocks;
    data.yBlocks = parser->yBlocks;
    data.redOutput = parser->redOutput;
    data.greenOutput = parser->greenOutput;
    data.blueOutput = parser->blueOutput;
    data.quantTable1 = parser->quantTable1;
    data.quantTable2 = parser->quantTable2;
    data.hf0codes = parser->hf0codes;
    data.hf1codes = parser->hf1codes;
    data.hf16codes = parser->hf16codes;
    data.hf17codes = parser->hf17codes;
    data.hf0lengths = parser->hf0lengths;
    data.hf1lengths = parser->hf1lengths;
    data.hf16lengths = parser->hf16lengths;
    data.hf17lengths = parser->hf17lengths;


    return data;
}

__global__ void processImagesKernel(JPEGParserData* deviceStructs, int numImages) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numImages) {
        // Process each image buffer (dummy operation, replace with decoding logic)
        // jpegParsers[idx].decode();
        // myKernel<<<1, 256>>>(idx);
        dim3 blockSize(8, 8);
        dim3 gridSize((deviceStructs[idx].width + blockSize.x - 1) / blockSize.x, (deviceStructs[idx].height + blockSize.y - 1) / blockSize.y);

        decodeKernel<<<gridSize, blockSize>>>(deviceStructs[idx].imageData, 
                                                deviceStructs[idx].luminous, 
                                                deviceStructs[idx].chromRed, 
                                                deviceStructs[idx].chromYel, 
                                                deviceStructs[idx].idctTable, 
                                                8, 8,  
                                                deviceStructs[idx].width, 
                                                deviceStructs[idx].height, 
                                                deviceStructs[idx].xBlocks, 
                                                deviceStructs[idx].yBlocks, 
                                                deviceStructs[idx].redOutput, 
                                                deviceStructs[idx].greenOutput, 
                                                deviceStructs[idx].blueOutput,
                                                deviceStructs[idx].quantTable1, 
                                                deviceStructs[idx].quantTable2, 
                                                deviceStructs[idx].hf0codes, 
                                                deviceStructs[idx].hf1codes, 
                                                deviceStructs[idx].hf16codes, 
                                                deviceStructs[idx].hf17codes, 
                                                deviceStructs[idx].hf0lengths, 
                                                deviceStructs[idx].hf1lengths, 
                                                deviceStructs[idx].hf16lengths, 
                                                deviceStructs[idx].hf17lengths
                                                );
    }
}

// Benchmark function for throughput measurement
void JPEGDecoderBenchmark(benchmark::State& state, std::vector<std::string> imagePaths) {

    size_t numImages = imagePaths.size();
    std::ofstream resultFile("benchmark_results.txt", std::ios_base::app);

    JPEGParser** jpegParsers = new JPEGParser*[numImages];
    JPEGParserData* structs = new JPEGParserData[numImages];

    // Create a new JPEGParser object for each image and store the pointer
    for (size_t i = 0; i < numImages; ++i) {
        jpegParsers[i] = new JPEGParser(imagePaths[i]);
        JPEGParser& parser = *jpegParsers[i];
        parser.extract();
        structs[i] = copyToStruct(jpegParsers[i]);
    }
    JPEGParserData* deviceStructs;
    hipMalloc(&deviceStructs, numImages * sizeof(JPEGParserData));
    hipMemcpy(deviceStructs, structs, numImages * sizeof(JPEGParserData), hipMemcpyHostToDevice);

    for (auto _ : state) {
        // Start timer
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Launch kernel
        int threadsPerBlock = 1;
        int numBlocks = numImages;
        // processImagesKernel<<<numBlocks, threadsPerBlock>>>(jpegParsers, numImages);
        processImagesKernel<<<numBlocks, threadsPerBlock>>>(deviceStructs, numImages);
        hipDeviceSynchronize();

        // Stop timer
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Calculate throughput
        double seconds = milliseconds / 1000.0;
        double throughput = numImages / seconds; // images per second
        double totalBytesProcessed = 0.0;
        for (const auto& path : imagePaths) {
            totalBytesProcessed += fs::file_size(path);  // Calculate total bytes processed
        }
        double bytesPerSecond = totalBytesProcessed / seconds; // bytes per second

        // Set iteration metrics
        state.SetIterationTime(seconds);
        state.counters["throughput_images_per_sec"] = throughput;
        state.counters["bytes_per_sec"] = bytesPerSecond;

        // Log results
        resultFile << "Throughput: " << throughput << " images/sec, "
                   << "Bytes per second: " << bytesPerSecond / (1024 * 1024) << " MB/sec\n";
    }
    resultFile.close();
    delete[] structs;
    // for (size_t i = 0; i < numImages; ++i) {
    //     delete jpegParsers[i]; // Delete each JPEGParser object
    // }
    // delete[] jpegParsers;
    hipFree(deviceStructs);
}

int main(int argc, char** argv) {
    std::string datasetPath = "/home/dphpc2024_jpeg_1/GPU-JPEG-Decoder/benchmarking_dataset";

    std::vector<std::string> imagePaths = getAllImages(datasetPath);

    if (imagePaths.empty()) {
        std::cout << "No images found in the dataset directory." << std::endl;
        return 1;
    }

    benchmark::RegisterBenchmark("BM_JPEGDecoder_Throughput", [imagePaths](benchmark::State& state) {
        JPEGDecoderBenchmark(state, imagePaths);
    })
    ->Unit(benchmark::kMillisecond)
    ->Iterations(10);

    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();

    return 0;
}
